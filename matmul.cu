#include "hip/hip_runtime.h"
#include "common.c"

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += (A.elements[row * A.stride + e] *
                   B.elements[e * B.stride + col]);
    C.elements[row * C.width + col] = Cvalue;
}


